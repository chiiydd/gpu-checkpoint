#include<cuda_original.h>

#include <cstdlib>
#include <cstring>
#include<stdio.h>
#include<communication.h>

#include<sys/socket.h>
#include<sys/un.h>
#include <unistd.h>

hipError_t proxy_call(CuDriverCallStructure *request,CuDriverCallReplyStructure * reply);
int proxy_init(){
    hipError_t result;
    const char* errorString;
    // 初始化 CUDA
    result=hipInit(0);
    if (result != hipSuccess){
        hipDrvGetErrorString(result, &errorString);

        printf("hipInit failed:%s\n",errorString);
        return 1;
    }
    hipCtx_t context;
    hipDevice_t cuDevice;
    hipDeviceGet(&cuDevice, 0);
    result=hipCtxCreate(&context, 0, cuDevice);

    if (result != hipSuccess){
        hipDrvGetErrorString(result, &errorString);
        printf("hipCtxCreate failed:%s\n",errorString);
        return -1;
    }
    return 0;
}

void proxy_start(){
    //建立socket server服务
    
    struct sockaddr_un sa_proxy;

    int skt_proxy,skt_accept;

    const char * socket_name=getenv("CUDA_PROXY_SOCKET");

    if (!socket_name){
        socket_name="CUDA_PROXY_SOCKET";
    }

    unlink(socket_name);

    memset(&sa_proxy, 0, sizeof(sa_proxy));

    strcpy(sa_proxy.sun_path, socket_name);
    sa_proxy.sun_family=AF_UNIX;

    if((skt_proxy=socket(AF_UNIX, SOCK_STREAM, 0))<0){
        perror("socket");
        exit(EXIT_FAILURE);
    }

    if(bind(skt_proxy, (struct sockaddr *)&sa_proxy, sizeof(sa_proxy))<0){
        perror("bind");
        exit(EXIT_FAILURE);
    }
    
    if (listen(skt_proxy, 5)<0){
        perror("listen");
        exit(EXIT_FAILURE);
    }

    if ((skt_accept=accept(skt_proxy, NULL, NULL))<0){
        perror("accept");
        exit(EXIT_FAILURE);
    }

    
    CuDriverCallStructure request;
    CuDriverCallReplyStructure reply;
    while(1){
        if(read(skt_accept, &request, sizeof(CuDriverCallStructure))<0){
            perror("read");
            exit(EXIT_FAILURE);
        }
        printf("op:%d\n",request.op);

        proxy_call(&request,&reply);

        write(skt_accept, &reply, sizeof(CuDriverCallReplyStructure));
        
    }

}

hipError_t proxy_call(CuDriverCallStructure *request,CuDriverCallReplyStructure * reply){
    
    switch (request->op) {
    
        case CuDriverCall::CuDriverGetVersion:
            reply->result=hipDriverGetVersion(&reply->returnParams.driverVersion);
            break;
        case CuDriverCall::CuInit:
                reply->result=hipInit(0);
            break;
        
        case CuDriverCall::CuGetExportTable:
            break;
        
        case CuDriverCall::CuModuleGetLoadingMode:
            reply->result=cuModuleGetLoadingMode(&reply->returnParams.mode);
        case  CuDriverCall::CuMemAlloc:
        case CuDriverCall::CuDeviceGetCount:
            reply->result=hipGetDeviceCount(&reply->returnParams.count);
            break;
        case CuDriverCall::CuMemcpyDtoH:
            break;
        case CuDriverCall::CuMemcpyHtoD:
            break;

        case CuDriverCall::CuCtxGetCurrent:
            break;
        case CuDriverCall::hipCtxPushCurrent:
            break;

        case CuDriverCall::CuDeviceGet:
            break;
        case CuDriverCall::CuDeviceGetAttribute:
            break;
        case CuDriverCall::CuDeviceGetName:
            break;
        case CuDriverCall::CuDeviceGetUuid:
            break;
        case CuDriverCall::hipDevicePrimaryCtxRelease:
            break;
        case CuDriverCall::CuDeviceTotalMem:
            break;
        case CuDriverCall::CuDevicePrimaryCtxRetain:
            break;

        
        
        default:
            break;
    }
    return hipSuccess;
}


int main(){
    proxy_init();
    proxy_start();
    return 0;
}

