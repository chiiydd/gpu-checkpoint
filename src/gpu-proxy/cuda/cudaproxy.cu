#include<hip/hip_runtime.h>

#include <cstdlib>
#include <cstring>
#include<stdio.h>
#include<communication.h>

#include<sys/socket.h>
#include<sys/un.h>
#include <unistd.h>

#include <string>

std::string to_string(CuDriverCall call) {
    switch (call) {
        case CuDriverCall::CuMemAlloc: return "CuMemAlloc";
        case CuDriverCall::CuMemFree: return "CuMemFree";
        case CuDriverCall::CuMemcpyHtoD: return "CuMemcpyHtoD";
        case CuDriverCall::CuMemcpyDtoH: return "CuMemcpyDtoH";
        case CuDriverCall::CuDriverGetVersion: return "CuDriverGetVersion";
        case CuDriverCall::CuDeviceGet: return "CuDeviceGet";
        case CuDriverCall::CuDeviceGetCount: return "CuDeviceGetCount";
        case CuDriverCall::CuDeviceGetName: return "CuDeviceGetName";
        case CuDriverCall::CuDeviceGetUuid: return "CuDeviceGetUuid";
        case CuDriverCall::CuDeviceTotalMem: return "CuDeviceTotalMem";
        case CuDriverCall::CuGetExportTable: return "CuGetExportTable";
        case CuDriverCall::CuModuleGetLoadingMode: return "CuModuleGetLoadingMode";
        case CuDriverCall::CuDeviceGetAttribute: return "CuDeviceGetAttribute";
        case CuDriverCall::CuCtxGetCurrent: return "CuCtxGetCurrent";
        case CuDriverCall::CuCtxSetCurrent: return "CuCtxSetCurrent";
        case CuDriverCall::CuDevicePrimaryCtxRetain: return "CuDevicePrimaryCtxRetain";
        case CuDriverCall::CuLibraryLoadData: return "CuLibraryLoadData";
        case CuDriverCall::CuLibraryUnload: return "CuLibraryUnload";
        case CuDriverCall::CuDevicePrimaryCtxRelease: return "CuDevicePrimaryCtxRelease";
        case CuDriverCall::CuCtxPushCurrent: return "CuCtxPushCurrent";
        case CuDriverCall::CuInit: return "CuInit";
        default: return "Unknown";
    }
}
hipError_t proxy_call(int socket_handle,CuDriverCallStructure *request,CuDriverCallReplyStructure * reply);
int proxy_init(){
    hipError_t result;
    const char* errorString;
    // 初始化 CUDA
    result=hipInit(0);
    if (result != hipSuccess){
        hipDrvGetErrorString(result, &errorString);

        printf("hipInit failed:%s\n",errorString);
        return 1;
    }
    hipCtx_t context;
    hipDevice_t cuDevice;
    hipDeviceGet(&cuDevice, 0);
    result=hipCtxCreate(&context, 0, cuDevice);

    if (result != hipSuccess){
        hipDrvGetErrorString(result, &errorString);
        printf("hipCtxCreate failed:%s\n",errorString);
        return -1;
    }
    return 0;
}

void proxy_start(){
    //建立socket server服务
    
    struct sockaddr_un sa_proxy;

    int skt_proxy,skt_accept;

    const char * socket_name=getenv("CUDA_PROXY_SOCKET");

    if (!socket_name){
        socket_name="CUDA_PROXY_SOCKET";
    }

    unlink(socket_name);

    memset(&sa_proxy, 0, sizeof(sa_proxy));

    strcpy(sa_proxy.sun_path, socket_name);
    sa_proxy.sun_family=AF_UNIX;

    if((skt_proxy=socket(AF_UNIX, SOCK_STREAM, 0))<0){
        perror("socket");
        exit(EXIT_FAILURE);
    }

    if(bind(skt_proxy, (struct sockaddr *)&sa_proxy, sizeof(sa_proxy))<0){
        perror("bind");
        exit(EXIT_FAILURE);
    }
    
    if (listen(skt_proxy, 100)<0){
        perror("listen");
        exit(EXIT_FAILURE);
    }


    
    CuDriverCallStructure request;
    CuDriverCallReplyStructure reply;
    while(1){
        if ((skt_accept=accept(skt_proxy, NULL, NULL))<0){
            perror("accept");
            continue;
        }

        if(read(skt_accept, &request, sizeof(CuDriverCallStructure))<0){
            perror("read");
            exit(EXIT_FAILURE);
        }
        printf("op:%s\n",to_string(request.op).c_str());

        proxy_call(skt_accept,&request,&reply);

        write(skt_accept, &reply, sizeof(CuDriverCallReplyStructure));
        close(skt_accept);
        
    }

}

hipError_t proxy_call(int socket_handle,CuDriverCallStructure *request,CuDriverCallReplyStructure * reply){
    
    void * buffer;
    switch (request->op) {
    
        case CuDriverCall::CuDriverGetVersion:
            reply->result=hipDriverGetVersion(&reply->returnParams.driverVersion);
            break;
        case CuDriverCall::CuInit:
                reply->result=hipInit(0);
            break;
        
        case CuDriverCall::CuGetExportTable:
            break;
        
        case CuDriverCall::CuModuleGetLoadingMode:
            reply->result=cuModuleGetLoadingMode(&reply->returnParams.mode);
        case  CuDriverCall::CuMemAlloc:

            reply->result=hipMalloc(&reply->returnParams.dptr, request->params.hipMalloc.size);
            break;
        case CuDriverCall::CuMemFree:
            reply->result=hipFree(request->params.hipFree.dptr);
            break;
        case CuDriverCall::CuDeviceGetCount:
            reply->result=hipGetDeviceCount(&reply->returnParams.count);
            break;
        case CuDriverCall::CuMemcpyHtoD:
            buffer=malloc(request->params.hipMemcpyHtoD.ByteCount);
            if(read(socket_handle,buffer,request->params.hipMemcpyHtoD.ByteCount)<0){
                perror("CuMemcpyDtoH:reading from cilent fails.\n");
                exit(EXIT_FAILURE);
            }
            reply->result=hipMemcpyHtoD(request->params.hipMemcpyHtoD.dstDevice, buffer, request->params.hipMemcpyHtoD.ByteCount);
            free(buffer);
            break;
        case CuDriverCall::CuMemcpyDtoH:
            buffer=malloc(request->params.hipMemcpyDtoH.ByteCount);
            reply->result=hipMemcpyDtoH(buffer, request->params.hipMemcpyDtoH.srcDevice, request->params.hipMemcpyDtoH.ByteCount);
            if(write(socket_handle,buffer,request->params.hipMemcpyDtoH.ByteCount)<0){
                perror("CuMemcpyDtoH:writing to cilent fails.\n");
            }
            free(buffer);
            break;

        case CuDriverCall::CuCtxGetCurrent:
            reply->result=hipCtxGetCurrent(&reply->returnParams.ctx);
        case CuDriverCall::CuCtxSetCurrent:
            reply->result=hipCtxSetCurrent(request->params.hipCtxSetCurrent.ctx);
            break;
        case CuDriverCall::CuCtxPushCurrent:
            reply->result=hipCtxPushCurrent(request->params.hipCtxPushCurrent.ctx);

            break;

        case CuDriverCall::CuDeviceGet:
            reply->result=hipDeviceGet(&reply->returnParams.device, request->params.hipDeviceGet.ordinal);
            break;
        case CuDriverCall::CuDeviceGetAttribute:
            reply->result=hipDeviceGetAttribute(&reply->returnParams.pi, request->params.hipDeviceGetAttribute.attrib, request->params.hipDeviceGetAttribute.dev);
            break;
        case CuDriverCall::CuDeviceGetName:
            break;
        case CuDriverCall::CuDeviceGetUuid:
            break;
        case CuDriverCall::CuDevicePrimaryCtxRelease:
            reply->result=hipDevicePrimaryCtxRelease(request->params.hipDevicePrimaryCtxRelease.dev);
            break;
        case CuDriverCall::CuDeviceTotalMem:
            reply->result=hipDeviceTotalMem(&reply->returnParams.bytes, request->params.hipDeviceTotalMem.dev);
            break;
        case CuDriverCall::CuDevicePrimaryCtxRetain:
            reply->result=hipDevicePrimaryCtxRetain(&reply->returnParams.ctx, request->params.hipDevicePrimaryCtxRetain.dev);
            break;

        case CuDriverCall::CuLibraryLoadData:
            
            break;
        
        default:
            break;
    }
    return hipSuccess;
}


int main(){
    proxy_init();
    proxy_start();
    return 0;
}

