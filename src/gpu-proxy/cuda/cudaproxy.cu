

#include <cstdlib>
#include <cstring>
#include<hip/hip_runtime.h>
#include<stdio.h>
#include<communication.h>

#include<sys/socket.h>
#include<sys/un.h>
#include <unistd.h>
const int PORT=8888;
int proxy_init(){
    hipError_t result;
    const char* errorString;
    // 初始化 CUDA
    result=hipInit(0);
    if (result != hipSuccess){
        hipDrvGetErrorString(result, &errorString);

        printf("hipInit failed:%s\n",errorString);
        return 1;
    }
    hipCtx_t context;
    hipDevice_t cuDevice;
    hipDeviceGet(&cuDevice, 0);
    result=hipCtxCreate(&context, 0, cuDevice);

    if (result != hipSuccess){
        hipDrvGetErrorString(result, &errorString);
        printf("hipCtxCreate failed:%s\n",errorString);
    }
}

void proxy_start(){
    //建立socket server服务
    
    struct sockaddr_un sa_proxy;

    int skt_proxy,skt_accept;

    const char * socket_name=getenv("CUDA_PROXY_SOCKET");

    if (!socket_name){
        socket_name="CUDA_PROXY_SOCKET";
    }

    unlink(socket_name);

    memset(&sa_proxy, 0, sizeof(sa_proxy));

    strcpy(sa_proxy.sun_path, socket_name);
    sa_proxy.sun_family=AF_UNIX;

    if((skt_proxy=socket(AF_UNIX, SOCK_STREAM, 0))<0){
        perror("socket");
        exit(EXIT_FAILURE);
    }

    if(bind(skt_proxy, (struct sockaddr *)&sa_proxy, sizeof(sa_proxy))<0){
        perror("bind");
        exit(EXIT_FAILURE);
    }
    
    if (listen(skt_proxy, 5)<0){
        perror("listen");
        exit(EXIT_FAILURE);
    }

    if ((skt_accept=accept(skt_proxy, NULL, NULL))<0){
        perror("accept");
        exit(EXIT_FAILURE);
    }

    
    cudaCallStructure structure;
    while(1){
        if(read(skt_accept, &structure, sizeof(cudaCallStructure))<0){
            perror("read");
            exit(EXIT_FAILURE);
        }
        
    }

}

hipError_t proxy_call(cudaCallStructure *structure){
    
    switch (structure->op) {
    
        case  CudaMalloc:
            break;

        case CudaMemcpy:
            break;
        case CudaFree:
            break;

        case CudaLaunchKernel:
            break;
        
        default:
            break;
    }
    return hipSuccess;
}


