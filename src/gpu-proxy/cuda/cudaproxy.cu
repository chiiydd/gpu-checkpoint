#include<hip/hip_runtime.h>
#include <cstdlib>
#include <cstring>
#include<stdio.h>
#include<communication.h>
#include<sys/socket.h>
#include<sys/un.h>
#include <unistd.h>

#include <string>

std::string to_string(CuDriverCall call) {
    switch (call) {
        case CuDriverCall::CuMemAlloc: return "CuMemAlloc";
        case CuDriverCall::CuMemFree: return "CuMemFree";
        case CuDriverCall::CuMemcpyHtoD: return "CuMemcpyHtoD";
        case CuDriverCall::CuMemcpyDtoH: return "CuMemcpyDtoH";
        case CuDriverCall::CuDriverGetVersion: return "CuDriverGetVersion";
        case CuDriverCall::CuDeviceGet: return "CuDeviceGet";
        case CuDriverCall::CuDeviceGetCount: return "CuDeviceGetCount";
        case CuDriverCall::CuDeviceGetName: return "CuDeviceGetName";
        case CuDriverCall::CuDeviceGetUuid: return "CuDeviceGetUuid";
        case CuDriverCall::CuDeviceTotalMem: return "CuDeviceTotalMem";
        case CuDriverCall::CuGetExportTable: return "CuGetExportTable";
        case CuDriverCall::CuModuleGetLoadingMode: return "CuModuleGetLoadingMode";
        case CuDriverCall::CuDeviceGetAttribute: return "CuDeviceGetAttribute";
        case CuDriverCall::CuCtxGetCurrent: return "CuCtxGetCurrent";
        case CuDriverCall::CuCtxSetCurrent: return "CuCtxSetCurrent";
        case CuDriverCall::CuDevicePrimaryCtxRetain: return "CuDevicePrimaryCtxRetain";
        case CuDriverCall::CuLibraryLoadData: return "CuLibraryLoadData";
        case CuDriverCall::CuLibraryUnload: return "CuLibraryUnload";
        case CuDriverCall::CuDevicePrimaryCtxRelease: return "CuDevicePrimaryCtxRelease";
        case CuDriverCall::CuCtxPushCurrent: return "CuCtxPushCurrent";
        case CuDriverCall::CuCtxPopCurrent: return "CuCtxPopCurrent";
        case CuDriverCall::CuInit: return "CuInit";
        case CuDriverCall::CuCtxCreate: return "CuCtxCreate";
        case CuDriverCall::CuLibraryGetModule: return "CuLibraryGetModule";
        case CuDriverCall::CuModuleGetFunction: return "CuModuleGetFunction";
        case CuDriverCall::CuLaunchKernel: return "CuLaunchKernel";
        default: return "Unknown";
    }
}
void   launch_kernel_proxy(CuDriverCallStructure * request,CuDriverCallReplyStructure * reply,void * parametersMetadata){
    void **cuda_args;
    uint16_t *arg_offsets;
    size_t param_num = *((size_t*)parametersMetadata);
    arg_offsets = (uint16_t*)(parametersMetadata+sizeof(size_t));
    cuda_args = (void **)malloc(param_num*sizeof(void*));
    for (size_t i = 0; i < param_num; ++i) {
        cuda_args[i] = parametersMetadata+sizeof(size_t)+param_num*sizeof(uint16_t)+arg_offsets[i];
    }

    reply->result = hipModuleLaunchKernel(request->params.hipModuleLaunchKernel.f, 
            request->params.hipModuleLaunchKernel.gridDimX,request->params.hipModuleLaunchKernel.gridDimY, request->params.hipModuleLaunchKernel.gridDimZ, 
            request->params.hipModuleLaunchKernel.blockDimX, request->params.hipModuleLaunchKernel.blockDimY, request->params.hipModuleLaunchKernel.blockDimZ,
            request->params.hipModuleLaunchKernel.sharedMemBytes, request->params.hipModuleLaunchKernel.hStream, cuda_args, NULL);
}
hipError_t proxy_call(int socket_handle,CuDriverCallStructure *request,CuDriverCallReplyStructure * reply);
int proxy_init(){
    // hipError_t result;
    // const char* errorString;
    // // 初始化 CUDA
    // result=hipInit(0);
    // if (result != hipSuccess){
    //     hipDrvGetErrorString(result, &errorString);

    //     printf("hipInit failed:%s\n",errorString);
    //     return 1;
    // }
    // hipCtx_t context;
    // hipDevice_t cuDevice;
    // hipDeviceGet(&cuDevice, 0);
    // result=hipCtxCreate(&context, 0, cuDevice);

    // if (result != hipSuccess){
    //     hipDrvGetErrorString(result, &errorString);
    //     printf("hipCtxCreate failed:%s\n",errorString);
    //     return -1;
    // }
    return 0;
}

void proxy_start(){
    //建立socket server服务
    
    struct sockaddr_un sa_proxy;

    int skt_proxy,skt_accept;

    const char * socket_name=getenv("CUDA_PROXY_SOCKET");

    if (!socket_name){
        socket_name="CUDA_PROXY_SOCKET";
    }

    unlink(socket_name);

    memset(&sa_proxy, 0, sizeof(sa_proxy));

    strcpy(sa_proxy.sun_path, socket_name);
    sa_proxy.sun_family=AF_UNIX;

    if((skt_proxy=socket(AF_UNIX, SOCK_STREAM, 0))<0){
        perror("socket");
        exit(EXIT_FAILURE);
    }

    if(bind(skt_proxy, (struct sockaddr *)&sa_proxy, sizeof(sa_proxy))<0){
        perror("bind");
        exit(EXIT_FAILURE);
    }
    
    if (listen(skt_proxy, 100)<0){
        perror("listen");
        exit(EXIT_FAILURE);
    }


    
    CuDriverCallStructure request;
    CuDriverCallReplyStructure reply;
    while(1){
        if ((skt_accept=accept(skt_proxy, NULL, NULL))<0){
            perror("accept");
            continue;
        }

        if(read(skt_accept, &request, sizeof(CuDriverCallStructure))<0){
            perror("read");
            exit(EXIT_FAILURE);
        }
        printf("op:%s\n",to_string(request.op).c_str());

        proxy_call(skt_accept,&request,&reply);

        write(skt_accept, &reply, sizeof(CuDriverCallReplyStructure));
        close(skt_accept);
        
    }

}

hipError_t proxy_call(int socket_handle,CuDriverCallStructure *request,CuDriverCallReplyStructure * reply){
    
    void * buffer;
    char * name;
    hipJitOption * jitOptions;
    void ** jitOptionValues;
    hipLibraryOption * libraryOptions;
    void ** libraryOptionValues;
    switch (request->op) {
    
        case CuDriverCall::CuDriverGetVersion:
            reply->result=hipDriverGetVersion(&reply->returnParams.driverVersion);
            break;
        case CuDriverCall::CuInit:
                reply->result=hipInit(request->params.hipInit.flags);
                printf("CuInit:%d\n",reply->result);
            break;
        
        case CuDriverCall::CuDeviceGetUuid:
                hipUUID uuid;
                reply->result=hipDeviceGetUuid(&uuid, request->params.hipDeviceGetUuid.dev);
                if(reply->result==hipSuccess){
                    memcpy(reply->returnParams.uuid,uuid.bytes,sizeof(uuid.bytes));
                }
            break;
        case CuDriverCall::CuGetExportTable:
            break;
        
        case CuDriverCall::CuModuleGetLoadingMode:
            reply->result=cuModuleGetLoadingMode(&reply->returnParams.mode);
            printf("cuModuleGetLoadingMode:%d\n",reply->returnParams.mode);
            printf("cuModuleGetLoadingMode result:%d\n",reply->result);
            break;
        case  CuDriverCall::CuMemAlloc:

            reply->result=hipMalloc(&reply->returnParams.dptr, request->params.hipMalloc.bytesize);
            break;
        case CuDriverCall::CuMemFree:
            reply->result=hipFree(request->params.hipFree.dptr);
            break;
        case CuDriverCall::CuDeviceGetCount:
            reply->result=hipGetDeviceCount(&reply->returnParams.count);
            break;
        case CuDriverCall::CuMemcpyHtoD:
            buffer=malloc(request->params.hipMemcpyHtoD.ByteCount);
            if(read(socket_handle,buffer,request->params.hipMemcpyHtoD.ByteCount)<0){
                perror("CuMemcpyDtoH:reading from cilent fails.\n");
                exit(EXIT_FAILURE);
            }
            reply->result=hipMemcpyHtoD(request->params.hipMemcpyHtoD.dstDevice, buffer, request->params.hipMemcpyHtoD.ByteCount);
            free(buffer);
            break;
        case CuDriverCall::CuMemcpyDtoH:
            buffer=malloc(request->params.hipMemcpyDtoH.ByteCount);
            reply->result=hipMemcpyDtoH(buffer, request->params.hipMemcpyDtoH.srcDevice, request->params.hipMemcpyDtoH.ByteCount);
            if(write(socket_handle,buffer,request->params.hipMemcpyDtoH.ByteCount)<0){
                perror("CuMemcpyDtoH:writing to cilent fails.\n");
            }
            free(buffer);
            break;

        case CuDriverCall::CuCtxCreate:
            reply->result=hipCtxCreate(&reply->returnParams.ctx,request->params.hipCtxCreate.flags,request->params.hipCtxCreate.dev);
        case CuDriverCall::CuCtxGetCurrent:
            reply->result=hipCtxGetCurrent(&reply->returnParams.ctx);
        case CuDriverCall::CuCtxSetCurrent:
            reply->result=hipCtxSetCurrent(request->params.hipCtxSetCurrent.ctx);
            break;
        case CuDriverCall::CuCtxPushCurrent:
            reply->result=hipCtxPushCurrent(request->params.hipCtxPushCurrent.ctx);

            break;
        case CUDriverCall::CuCtxPopCurrent:
            reply->result=hipCtxPopCurrent(&reply->returnParams.ctx);
            break;
        case CuDriverCall::CuDeviceGet:
            reply->result=hipDeviceGet(&reply->returnParams.device, request->params.hipDeviceGet.ordinal);
            break;
        case CuDriverCall::CuDeviceGetAttribute:
            reply->result=hipDeviceGetAttribute(&reply->returnParams.pi, request->params.hipDeviceGetAttribute.attrib, request->params.hipDeviceGetAttribute.dev);
            break;
        case CuDriverCall::CuDeviceGetName:
            buffer=malloc(request->params.hipDeviceGetName.len);
            reply->result=hipDeviceGetName((char *)buffer, request->params.hipDeviceGetName.len, request->params.hipDeviceGetName.device);
            if(write(socket_handle, buffer, request->params.hipDeviceGetName.len)<0){
                perror("CuDeviceGetName:writing to cilent fails.\n");
            }
            free(buffer);
            break;

        case CuDriverCall::CuDevicePrimaryCtxRelease:
            reply->result=hipDevicePrimaryCtxRelease(request->params.hipDevicePrimaryCtxRelease.dev);
            break;
        case CuDriverCall::CuDeviceTotalMem:
            reply->result=hipDeviceTotalMem(&reply->returnParams.bytes, request->params.hipDeviceTotalMem.dev);
            break;
        case CuDriverCall::CuDevicePrimaryCtxRetain:
            reply->result=hipDevicePrimaryCtxRetain(&reply->returnParams.ctx, request->params.hipDevicePrimaryCtxRetain.dev);
            break;

        case CuDriverCall::CuLibraryLoadData:
            if(request->params.hipLibraryLoadData.numJitOptions!=0){
                jitOptions=(hipJitOption *)malloc(sizeof(hipJitOption)*request->params.hipLibraryLoadData.numJitOptions);
                if(read(socket_handle,jitOptions,sizeof(hipJitOption)*request->params.hipLibraryLoadData.numJitOptions)<0){
                    perror("CuLibraryLoadData:reading from cilent fails.\n");
                    exit(EXIT_FAILURE);
                }
                jitOptionValues=(void **)malloc(sizeof(void *)*request->params.hipLibraryLoadData.numJitOptions);
                for(int i=0;i<request->params.hipLibraryLoadData.numJitOptions;i++){
                    jitOptionValues[i]=malloc(sizeof(int));
                    if(read(socket_handle,jitOptionValues[i],sizeof(int))<0){
                        perror("CuLibraryLoadData:reading from cilent fails.\n");
                        exit(EXIT_FAILURE);
                    }
                }
            }
            if(request->params.hipLibraryLoadData.numLibraryOptions!=0){
                libraryOptions=(hipLibraryOption *)malloc(sizeof(hipLibraryOption)*request->params.hipLibraryLoadData.numLibraryOptions);
                if(read(socket_handle,libraryOptions,sizeof(hipJitOption)*request->params.hipLibraryLoadData.numLibraryOptions)<0){
                    perror("CuLibraryLoadData:reading from cilent fails.\n");
                    exit(EXIT_FAILURE);
                }
                libraryOptionValues=(void **)malloc(sizeof(void *)*request->params.hipLibraryLoadData.numLibraryOptions);
                for(int i=0;i<request->params.hipLibraryLoadData.numLibraryOptions;i++){
                    libraryOptionValues[i]=malloc(sizeof(int));
                    if(read(socket_handle,libraryOptionValues[i],sizeof(int))<0){
                        perror("CuLibraryLoadData:reading from cilent fails.\n");
                        exit(EXIT_FAILURE);
                    }
                }
            }
            buffer=malloc(request->params.hipLibraryLoadData.fatbinSize);
            if(read(socket_handle,buffer,request->params.hipLibraryLoadData.fatbinSize)<0){
                perror("CuLibraryLoadData:reading fat binary from cilent fails.\n");
                exit(EXIT_FAILURE);
            }
            reply->result=hipLibraryLoadData(&reply->returnParams.lib,buffer,jitOptions,jitOptionValues,request->params.hipLibraryLoadData.numJitOptions,libraryOptions,libraryOptionValues,request->params.hipLibraryLoadData.numLibraryOptions);
            free(buffer);
            break;
        case CuDriverCall::CuLaunchKernel:
            if(read(socket_handle,buffer,request->params.hipModuleLaunchKernel.parametersMetadataLen)<0){
                perror("CuLaunchKernel:reading parametersMetadata from cilent fails.\n");
                exit(EXIT_FAILURE);
            }
            launch_kernel_proxy(request, reply, buffer);
            break;
        case CuDriverCall::CuLibraryGetModule:
            reply->result= cuLibraryGetModule(&reply->returnParams.mod,request->params.cuLibraryGetModule.library);
            break;
        case CuDriverCall::CuModuleGetFunction:
            name=(char *)malloc(request->params.hipModuleGetFunction.nameLength);
            if(read(socket_handle, name, request->params.hipModuleGetFunction.nameLength)<0){
                perror("CuModuleGetFunction:reading from cilent fails.\n");
                exit(EXIT_FAILURE);
            }
            reply->result=hipModuleGetFunction(&reply->returnParams.hfunc,request->params.hipModuleGetFunction.mod,request->params.hipModuleGetFunction.name);
            free(name);
            break;
        default:
            break;
    }
    return hipSuccess;
}


int main(){
    proxy_init();
    proxy_start();
    return 0;
}

