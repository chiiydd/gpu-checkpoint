#include "hip/hip_runtime.h"
#include <cstddef>
#include <cstdint>
#include <cstdio>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <iostream>

void hexdump(const uint8_t* data, size_t size)
{
    size_t pos = 0;
    while (pos < size) {
        printf("%#05zx: ", pos);
        for (int i = 0; i < 16; i++) {
            if (pos + i < size) {
                printf("%02x", data[pos + i]);
            } else {
                printf("  ");
            }
            if (i % 4 == 3) {
                printf(" ");
            }
        }
        printf(" | ");
        for (int i = 0; i < 16; i++) {
            if (pos + i < size) {
                if (data[pos + i] >= 0x20 && data[pos + i] <= 0x7e) {
                    printf("%c", data[pos + i]);
                } else {
                    printf(".");
                }
            } else {
                printf(" ");
            }
        }
        printf("\n");
        pos += 16;
    }
}
extern "C" {
  /*
  * These defines are for the fatbin.c runtime wrapper
  */
  #define FATBINC_MAGIC   0x466243B1
  #define FATBINC_VERSION 1
  #define FATBINC_LINK_VERSION 2

  typedef struct {
    int magic;
    int version;
    const unsigned long long* data;
    void *filename_or_fatbins;  /* version 1: offline filename,
                                * version 2: array of prelinked fatbins */
  }FatBinaryWrapper;
}
struct  alignas(8) FatBinaryHeader     
{
    uint32_t 			magic;
    uint16_t         	version;
    uint16_t         	headerSize;
    uint64_t 	fatSize;
};


struct  alignas(8) FatEntryHeader
{
    uint16_t kind; 
    uint16_t unknown1;
    uint32_t headerSize;
    uint64_t size;
    uint32_t compressedSize;       // Size of compressed data
    uint32_t unknown2;              // Address size for PTX?
    uint16_t minor;
    uint16_t major;
    uint32_t arch;
    uint32_t objNameOffset;
    uint32_t objNameLen;
    uint64_t flags;
    uint64_t zero;                  // Alignment for compression?
    uint64_t decompressedSize;     // Length of compressed data in decompressed representation.
                                    // There is an uncompressed footer so this is generally smaller
                                    // than size.
};
__global__ void add(int *dev_a, int *dev_b, int *dev_c) {
    dev_c[threadIdx.x] = dev_a[threadIdx.x] + dev_b[threadIdx.x];
}

void cudart_call(){
    int a[5] = {1, 2, 3, 4, 5};
    int b[5] = {10, 20, 30, 40, 50};
    int c[5] = {0, 0, 0, 0, 0};

    int *dev_a, *dev_b, *dev_c;
    hipMalloc((void**)&dev_a, 5 * sizeof(int));
    hipMalloc((void**)&dev_b, 5 * sizeof(int));
    hipMalloc((void**)&dev_c, 5 * sizeof(int));

    hipMemcpy(dev_a, a, 5 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, 5 * sizeof(int), hipMemcpyHostToDevice);

    add<<<1, 5>>>(dev_a, dev_b, dev_c);

    hipMemcpy(c, dev_c, 5 * sizeof(int), hipMemcpyDeviceToHost);

    for(int i = 0; i < 5; i++){
        std::cout << c[i] << std::endl;
    }

    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);

}

void print_cuda_error(hipError_t result){
    const char * error_string;
    hipDrvGetErrorString(result, &error_string);
    std::cout<<"CUDA Error: "<<error_string<<std::endl;
}

void * read_file(const char * filename, size_t * size){
    FILE * file = fopen(filename, "rb");
    if(!file){
        return NULL;
    }
    
    void * buffer;
    int result;
    result=fseek(file, 0, SEEK_END);
    if(result == -1){
        std::cout<<"Error in fseek"<<std::endl;
        fclose(file);
        return NULL;
    }
    long file_size = ftell(file);
    if(file_size == -1){
        std::cout<<"Error in ftell"<<std::endl;
        fclose(file);
        return NULL;
    }
    *size = file_size;
    
    buffer=malloc(file_size);
    rewind(file); // 将文件指针重置到文件开头


    result = fread(buffer, 1, file_size, file);
    if(result != file_size){
        std::cout<<"Error in fread"<<std::endl;
        fclose(file);
        free(buffer);
        return NULL;
    }
    fclose(file);

    
    // hexdump((uint8_t *)buffer, file_size);
    return buffer;
}
void cuda_call(){
    int a[5] ={1,2,3,4,5};
    int b[5] ={10,20,30,40,50};
    int c[5] ={0,0,0,0,0};
    
    hipDevice_t cuDevice;
    hipCtx_t cuContext;
    CUmoduleLoadingMode mode;
    hipError_t result;
    hipLibrary_t library;
    hipModule_t mod;
    hipFunction_t func;

    hipInit(0);
    hipDeviceGet(&cuDevice, 0);
    std::cout<<"Device: "<<cuDevice<<std::endl;

    char * gpu_name = new char[100];
    hipDeviceGetName(gpu_name, 100, cuDevice);
    std::cout<<"GPU Name: "<<gpu_name<<std::endl;


    result=cuModuleGetLoadingMode(&mode);
    if (result!=hipSuccess){
        std::cout<<"[cuModuleGetLoadingMode] fails\n";
        print_cuda_error(result);
        return;
    }

    std::cout<<"[cuModuleGetLoadingMode] get mod:"<<mode<<std::endl;
    hipCtxCreate(&cuContext, 0, cuDevice);
    std::cout<<"[hipCtxCreate] Context Create: "<<cuContext<<std::endl;

    hipCtx_t ctx;
    hipCtxGetCurrent(&ctx);
    std::cout<<"[hipCtxGetCurrent] Current Context: "<<ctx<<std::endl;

    hipCtxSetCurrent(cuContext);

    hipDevicePrimaryCtxRetain(&ctx, cuDevice);

    std::cout<<"[hipDevicePrimaryCtxRetain] Primary Context: "<<ctx<<std::endl;
    hipDevice_t dev;
    hipCtxGetDevice(&dev);
    std::cout<<"[hipCtxGetDevice] Current Context Device: "<<dev<<std::endl;

    int *dev_a, *dev_b, *dev_c;
    hipMalloc((hipDeviceptr_t*)&dev_a, 5 * sizeof(int));
    hipMalloc((hipDeviceptr_t*)&dev_b, 5 * sizeof(int));
    hipMalloc((hipDeviceptr_t*)&dev_c, 5 * sizeof(int));

    hipMemcpyHtoD((hipDeviceptr_t)dev_a, a, 5 * sizeof(int));
    hipMemcpyHtoD((hipDeviceptr_t)dev_b, b, 5 * sizeof(int));



    void * fatbin;
    size_t size;
    fatbin=read_file("/workspace/test.fatbin", &size);
    if(fatbin == NULL){
        std::cout<<"Error in read  fatbinary file"<<std::endl;
        return;
    }

    std::cout<<"Fatbin address: "<<&fatbin<<std::endl;
    FatBinaryHeader *header = (FatBinaryHeader*)fatbin;
    
    //打印fatbin头部信息
    std::cout<<"Magic: "<<std::hex<<header->magic<<std::endl;

    //取消十六进制输出
    std::cout<<std::dec;

    std::cout<<"Version: "<<header->version<<std::endl;
    std::cout<<"Header Size: "<<header->headerSize<<std::endl;
    std::cout<<"Fat Size: "<<header->fatSize<<std::endl;


	FatEntryHeader * fatbinEntry=(FatEntryHeader *) ((char *)fatbin+header->headerSize);
	
    //打印fatbinEntry头部信息
    std::cout<<"Kind: "<<fatbinEntry->kind<<std::endl;
    std::cout<<"Unknown1: "<<fatbinEntry->unknown1<<std::endl;
    std::cout<<"Header Size: "<<fatbinEntry->headerSize<<std::endl;
    std::cout<<"Size: "<<fatbinEntry->size<<std::endl;
    std::cout<<"Compressed Size: "<<fatbinEntry->compressedSize<<std::endl;
    std::cout<<"Unknown2: "<<fatbinEntry->unknown2<<std::endl;
    std::cout<<"Minor: "<<fatbinEntry->minor<<std::endl;
    std::cout<<"Major: "<<fatbinEntry->major<<std::endl;
    std::cout<<"Arch: "<<fatbinEntry->arch<<std::endl;

    FatBinaryWrapper wrapper ={FATBINC_MAGIC, FATBINC_LINK_VERSION, (const unsigned long long*)fatbin, NULL};

    hipLibraryOption option[]={
        hipLibraryBinaryIsPreserved
    };
    
    int values []={1};
    result= hipLibraryLoadData(&library, &wrapper, nullptr , nullptr, 0, option,(void **)&values ,1);

    std::cout<<"[hipLibraryLoadData] get lib:"<<library<<std::endl;
    if(result != hipSuccess){

        std::cout<<"Error in hipLibraryLoadData"<<std::endl;
        print_cuda_error(result);
        return;
    }
    hipCtxGetCurrent(&ctx);
    std::cout<<"[hipCtxPushCurrent] Current Context: "<<ctx<<std::endl;

    result=hipCtxPushCurrent(ctx);


    if(result != hipSuccess){

        std::cout<<"Error in hipCtxPushCurrent"<<std::endl;
        print_cuda_error(result);
        return;
    }
    
    hipCtxGetCurrent(&ctx);
    std::cout<<"[hipCtxPushCurrent] Current Context: "<<ctx<<std::endl;


    result =cuLibraryGetModule(&mod, library);
    if(result != hipSuccess){

        std::cout<<"Error in cuLibraryGetModule:"<<result<<std::endl;
        print_cuda_error(result);

        return;
    }

    result =hipCtxPopCurrent(&ctx);
    result = hipModuleGetFunction(&func, mod, "_Z3addPiS_S_");
    if(result != hipSuccess){
        std::cout<<"Error in hipModuleGetFunction"<<std::endl;
        print_cuda_error(result);

        return;
    }

    void *args[] = {&dev_a, &dev_b, &dev_c};
    result = hipModuleLaunchKernel(func, 1, 1, 1, 5, 1, 1, 0, 0, args, 0);
    if(result != hipSuccess){
        std::cout<<"Error in hipModuleLaunchKernel"<<std::endl;
        return;
    }
    
    hipMemcpyDtoH(c, (hipDeviceptr_t)dev_c, 5 * sizeof(int));

    for(int i = 0; i < 5; i++){
        std::cout << c[i] << std::endl;
    }
    hipFree((hipDeviceptr_t)dev_a);
    hipFree((hipDeviceptr_t)dev_b);
    hipFree((hipDeviceptr_t)dev_c);
}
int main(){
    // int device_count;
    // hipGetDeviceCount(&device_count);
    // for (int i = 0; i < device_count; i++) {
    //     hipDeviceProp_t prop;
    //     hipGetDeviceProperties(&prop, i);

    //     printf("Device %d: %s\n", i, prop.name);
    //     printf("  Compute Capability: %d.%d\n", prop.major, prop.minor);
    // }
    cuda_call();

    // cudart_call();
    return 0;
}